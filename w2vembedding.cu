#include "hip/hip_runtime.h"
#include "w2vembedding.hh"
#include <iostream>

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

using namespace std;

// outOffset es por facilidad
__global__ void updateGrads(float* dictionary, float* grad_center, float *grad_outside, int *d_idx, int embed_size, int outOffset, int low_bound, int batch_size, int sents_num)
{
    int fil = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if(col < embed_size)
    {
        if(fil < batch_size)
        {

            if(fil != low_bound) 
            {
                dictionary[(d_idx[fil]*embed_size+col)+outOffset] +=  grad_outside[fil*embed_size+col]/sents_num;
            }

            else                    // caso center - grad_center ~ (1, embed_size)
            {
                dictionary[d_idx[fil]*embed_size+col+outOffset] += grad_center[col]/sents_num;
            

            }

            
        }
    }
}

W2VEmbedding::W2VEmbedding(Shape W2V_shape, int contextSize, int sents_num):
    cost(W2V_shape.y, contextSize), W2V(2*W2V_shape.x, W2V_shape.y)
{   
    W2V.allocateMemory();
    initW2V();

    this -> context = contextSize;  // tamaño de ventana. alrededor de una palabra central, se toman a lo sumo 2*context palabras
    this -> sents_num = sents_num;
    this -> outOffset = (W2V.shape.x/2)*W2V.shape.y;
    
    hipMalloc(&d_centerVec, W2V.shape.y*sizeof(float));    // (1, embed_size)
    hipMalloc(&d_outsideVecs, 2*context*W2V.shape.y*sizeof(float));    // (context, embed_size)
    hipMalloc(&d_idx, 2*context*sizeof(int));

    hipMemset(d_centerVec, 0, W2V.shape.y*sizeof(float));
    hipMemset(d_outsideVecs, 0, 2*context*W2V.shape.y*sizeof(float));
    hipMemset(d_idx, 0, 2*context*sizeof(int));

}

W2VEmbedding::~W2VEmbedding()
{
    hipFree(d_centerVec);
    hipFree(d_outsideVecs);
    hipFree(d_idx);
}

void W2VEmbedding::initW2V()
{
    // genero vocab_size*embed_size aleatorios y vocab_size*embed_size seteados en ceros : Center y OutsideVectors
    hiprandGenerator_t prgen;
    float *deviceOutside = W2V.data_d.get()+W2V.shape.x/2*W2V.shape.y;

    cout << "Create Generator : " << hiprandCreateGenerator(&prgen, HIPRAND_RNG_PSEUDO_DEFAULT)<< endl;
    cout << "Seed : " << hiprandSetPseudoRandomGeneratorSeed(prgen, 12314ULL) << endl;

    cout << "Generate Normal : " << hiprandGenerateNormal(prgen, W2V.data_d.get(), W2V.shape.x*W2V.shape.y, 0, 1.0f/W2V.shape.y)<< endl;

    cout << "Destroy : " << hiprandDestroyGenerator(prgen)<< endl;

    // genero vocab_size*embed_size ceros
    cout << "Memset : " << hipMemset(deviceOutside, 0, W2V.shape.x/2*W2V.shape.y*sizeof(float)) << endl;
}
// centerIdx es la dirección de memoria que apunta al vector central

void W2VEmbedding::updateDictionary(int *h_Idx, int sentID, int cWordID, int low_bound, int up_bound)
{   
    
    int batch_size = (up_bound-low_bound);
    int j = 0;
    assert(batch_size>0);

    // for(int i=0; i<up_bound; i++)
    // {
    //     cout<< centerIdx[i] << '\n';
    // }
    // vector de posiciones en diccionario - copio lo que obtengo para una palabra -- esto podría ser modificado
    hipMemcpy(d_idx, &h_Idx[sentID+low_bound], batch_size, hipMemcpyHostToDevice);

    // copio memoria del vector central
    hipMemcpy(d_centerVec, &W2V.data_d.get()[W2V.shape.y*h_Idx[sentID+cWordID]], W2V.shape.y*sizeof(float), hipMemcpyDeviceToDevice);

    // Mapeo a vectores
    for(int idx = low_bound; idx <= up_bound; idx++)
    {
        if (idx == cWordID) continue;  // la central la omito
        
        // agarro el vector correspondiente a cada palabra en outsideVectors y la voy copiando

        hipMemcpy(&d_outsideVecs[j*W2V.shape.y], &W2V.data_d.get()[outOffset+h_Idx[sentID + idx]*W2V.shape.y], W2V.shape.y*sizeof(float), hipMemcpyDeviceToDevice);
        j++;
    }

    // Hasta acá, chequeado
    
    // calculo las actualizaciones para la palabra central y las de los costados
    // sobre d_centerVec y d_outsideVecs 
    cost.lossAndGrad(d_centerVec, d_outsideVecs, batch_size);
    
    // kernet bidimensional para actualizar matriz
    dim3 block_size(256, 256);
    dim3 block_num((batch_size+block_size.x-1)/block_size.x, (W2V.shape.y+block_size.y-1)/block_size.y);

    updateGrads<<<block_num, block_size>>>(W2V.data_d.get(), d_centerVec, d_outsideVecs, d_idx, W2V.shape.y, outOffset, low_bound, batch_size, sents_num);
}
#include "hip/hip_runtime.h"
#include "costfun.hh"

using namespace std;
using namespace thrust::placeholders;

void print_matrix(double *Mat, int Mat_height, int Mat_width){

    double* aux = (double *)malloc(sizeof(double)*Mat_width*Mat_height);

    hipMemcpy(aux, Mat, sizeof(double)*Mat_width*Mat_height, hipMemcpyDeviceToHost);

    printf("fil : %d, col : %d\n", Mat_width, Mat_height);
    for(int i=0; i<Mat_height; i++)
    {
        for(int j=0; j<Mat_width; j++)
        {
            printf("%.10f ",aux[i*Mat_width+j]);
        }
        printf("\n");
    }

    free(aux);
}

// non safe at all
__global__ void logitsSoftmax(double *wordVecs, double *Y_est, int centerIdx, int vocab_size, int embed_size, int offset)
{
    // para cada fila tomo los indices del thread 
    int fil = blockIdx.x * blockDim.x + threadIdx.x;

    double logits_value = 0.0;

    if (fil < vocab_size)
    {   
        for (int i=0 ; i < embed_size; i++)
        {   
            // recorro las filas de Offset vectors
            logits_value +=  wordVecs[offset+fil*embed_size+i]*wordVecs[centerIdx*embed_size+i];   
        }  
        
        Y_est[fil] = exp2(logits_value);
    }
}
// gradiente con respecto a la palabra clave (ya le paso el softmax)
// transpongo la matriz de palabras así le actualizo todo
__global__ void gradCenterVec(double* outsideVecs, double* Y_est, double *gradCenter,  int vocab_size, int embed_size)
{   
    int fil = blockIdx.x * blockDim.x + threadIdx.x;

    double grad = 0.0;

    if (fil<embed_size)
    {
        for (int i=0 ; i < vocab_size; i++)
        {
            grad += outsideVecs[i*embed_size+fil]*Y_est[i];
        }
        gradCenter[fil] += grad;
    }
    __syncthreads();
}

// hago producto externo entre center vecs y y-y_est para actualizar palabras outside
__global__ void gradOutsideVecs(double *centerVec, double *Y_est, double *gradOutside,  int vocab_size, int embed_size)
{
    int fil = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(fil < vocab_size)
    { 
        if (col < embed_size)
        {
            gradOutside[fil*embed_size+col] += Y_est[fil]*centerVec[col];   
        }
    }
    __syncthreads();
}

// update implica Y = Y_est - Y
__global__ void updateY(double *Y, double *loss, int* out_idxs, int currIdx)
{   
    Y[out_idxs[currIdx]] += -1;
    *loss += log2(Y[out_idxs[currIdx]]);
    __syncthreads();
}

__global__ void upCenter(double *centerVec, double *grad_center, double lr, int embed_size, int  batch_size)
{
    int fil = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(fil < embed_size)
    {   
        centerVec[fil] -= lr*grad_center[fil]/batch_size;   
    }
    __syncthreads();
}

__global__ void upOutside(double *outsideVecs, double *grad_outside, double lr, int embed_size, int vocab_size, int batch_size)
{   
    int fil = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(fil < embed_size)
    {   
        if(col < vocab_size)
        {   
            outsideVecs[col*embed_size + fil] -= lr*grad_outside[col*embed_size + fil]/batch_size;   
        }
    }
    __syncthreads();
}

// le paso el vector central y los vectores outside 
// vec central es (embed_size, 1), vec outside es (k, embed_size)
// https://devblogs.nvidia.com/unified-memory-cuda-beginners/ por hipMallocManaged
// vec central YA VIENE TRANSPUESTO, ver si es una decision piola o lo transpongo en kernel, c'est le meme
// agarro cada uno de los logits, los exponencio y obtengo una densidad de probabilidad
// de cada palabra externa dada una central
// cost es un vector de K elementos que me da una probabilidad empírica de lo cercanas que estan dos palabras en este espacio. es en el mismo sentido, la entropia conjunta entre la palabra real y_i {i=1,...,k}(con prob 1) y la palabra predicha y^{\hat}_i {i=1,...,k}

W2VCost::W2VCost(int embed_size, int vocab_size, double lr)
{
    // el máximo que voy a requerir es context
    this -> embed_size = embed_size;
    this -> vocab_size = vocab_size;
    this -> out_offset = vocab_size*embed_size;

    this -> lr = lr;
    this -> iteration = 0;

    hipMalloc(&Y_est, vocab_size*sizeof(double));
    hipMalloc(&grad_center, embed_size*sizeof(double));    // (1, embed_size)
    hipMalloc(&grad_outside, vocab_size*embed_size*sizeof(double));    // (context, embed_size)
    hipMalloc(&loss, sizeof(double));

    hipMemset(Y_est, 0, vocab_size*sizeof(double));
    hipMemset(grad_center, 0,  embed_size*sizeof(double));
    hipMemset(grad_outside, 0,  vocab_size*embed_size*sizeof(double));
    hipMemset(loss, 0,  sizeof(double));
}

W2VCost::~W2VCost()
{
    hipFree(this -> grad_center);
    hipFree(this -> grad_outside);
    hipFree(this -> loss);
    hipFree(this -> Y_est);
}

// para cada palabra externa
void W2VCost::lossAndGrad(double* wordVecs, int* outsideIdxs,  int centerIdx, int context_size)
{      
    // double *aux = (double*)malloc(sizeof(double)*vocab_size*embed_size);
    // por cada palabra del contexto, actualizo
    for(int currentOutIdx=0; currentOutIdx<context_size; currentOutIdx++)
    {   
        W2VCost::softLoss(wordVecs, centerIdx);
        updateY<<<1,1>>>(Y_est, loss, outsideIdxs, currentOutIdx);
        // // actualizo gradientes 
        W2VCost::gradCenter(&wordVecs[out_offset]);
        W2VCost::gradOutside(&wordVecs[centerIdx*embed_size]);

        hipMemset(Y_est, 0,  vocab_size*sizeof(double));
        gpuErrchk(hipPeekAtLastError());
    }
}

void W2VCost::updateGradients(double* wordVecs, int centerIdx)
{   
    
    updateCenter(&wordVecs[embed_size*centerIdx]);
    updateOutside(&wordVecs[out_offset]);
    
    this -> iteration ++;
    if((this -> iteration%this->batch_size) == 0) 
    {
        this -> lr *= 0.5;
    }

    cout << this->iteration << endl;

    hipMemset(grad_center, 0,  embed_size*sizeof(double));
    gpuErrchk(hipPeekAtLastError());

    hipMemset(grad_outside, 0,  vocab_size*embed_size*sizeof(double));
    gpuErrchk(hipPeekAtLastError());
}

void W2VCost::updateCenter(double* centerVec)
{
        // necesito vocab_size threads
    dim3 block_size(256);
    dim3 block_num((embed_size+block_size.x-1)/block_size.x);

    upCenter<<<block_num, block_size>>>(centerVec, grad_center, lr, embed_size, batch_size);
    gpuErrchk(hipPeekAtLastError());
}

void W2VCost::updateOutside(double* outsideVecs)
{
    // necesito vocab_size threads
    dim3 block_size(8, 8);
    dim3 block_num((embed_size+block_size.x-1)/block_size.x, (vocab_size+block_size.y-1)/block_size.y);

    upOutside<<<block_num, block_size>>>(outsideVecs, grad_outside, lr, embed_size, vocab_size, batch_size);
    gpuErrchk(hipPeekAtLastError());
}

void W2VCost::softLoss(double *wordVecs, int centerVecIdx)
{   
    double sum = 0.0;
    
    // necesito vocab_size threads
    dim3 block_size(256);
    dim3 block_num((vocab_size+block_size.x-1)/block_size.x);

    assert(out_offset == vocab_size*embed_size);
    assert(centerVecIdx < vocab_size);

    // print_matrix(wordVecs, 2*vocab_size, embed_size);
    // hago los k productos punto entre central y las outside
    logitsSoftmax<<<block_num, block_size>>>(wordVecs, Y_est, centerVecIdx, vocab_size, embed_size, out_offset);
    gpuErrchk(hipPeekAtLastError());

    thrust::device_ptr<double>Y_dev = thrust::device_pointer_cast(Y_est);
    sum = thrust::reduce(Y_dev, Y_dev+vocab_size, 0, thrust::plus<double>()); 
    // // acá realmente hago softmax
    thrust::transform(Y_dev, Y_dev+vocab_size, Y_dev, _1/sum);
    gpuErrchk(hipPeekAtLastError());
 
    // *loss = -logf(*loss); 
}

void W2VCost::gradCenter(double *outsideVecs)
{   
    // cout <<"Batch size gradcen : " << batch_size << endl;
    // necesito embed_size threads
    dim3 block_size(256);
    dim3 block_num((embed_size+block_size.x-1)/block_size.x);

    gradCenterVec<<<block_num, block_size>>>(outsideVecs, Y_est, grad_center, vocab_size, embed_size);
    gpuErrchk(hipPeekAtLastError());
}

void W2VCost::gradOutside(double *centerVec)
{
    // cout <<"Batch size gradout : " << batch_size << endl;
    dim3 block_size(8, 8);
    dim3 block_num((vocab_size+block_size.x-1)/block_size.x, (embed_size+block_size.y-1)/block_size.y);
    
    // cout << "Antes" << endl;
    // print_matrix(grad_outside, vocab_size, embed_size);
    gradOutsideVecs<<<block_num, block_size>>>(centerVec, Y_est, grad_outside, vocab_size, embed_size);
    gpuErrchk(hipPeekAtLastError());
}